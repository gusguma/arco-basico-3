#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
/// PROGRAMACIÓN EN CUDA C/C++
/// Práctica:	BASICO 3 : Hilos y Bloques
/// Autor:		Gustavo Gutierrez Martin
/// Fecha:		Octubre 2022
///////////////////////////////////////////////////////////////////////////

/// dependencias ///
#include <cstdio>
#include <cstdlib>
#include <ctime>

/// constantes ///
#define MB (1<<20) /// MiB = 2^20
#define MAX_THREADS_PER_BLOCK 10
#define MAX_INT_NUMBER 2147483647

/// muestra por consola que no se ha encontrado un dispositivo CUDA
int getErrorDevice();
/// muestra los datos de los dispositivos CUDA encontrados
int getDataDevice(int deviceCount);
/// numero de CUDA cores
int getCudaCores(hipDeviceProp_t deviceProperties);
/// muestra por pantalla las propiedades del dispositivo CUDA
int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties);
/// solicita al usuario el número de elementos que se sumaran
int requestNumberOfItems(int *numberOfItems);
/// inicializa el array del host
int loadHostData(int *hst_vector1, int *hst_vector2, int numberOfItems);
/// transfiere los datos del host al device
int dataTransferToDevice(int *hst_vector1, int *dev_vector1, int numberOfItems);
/// realiza la suma de los arrays en el device
__global__ void suma(const int *dev_vector1, int *dev_vector2, int *dev_result, int numberOfItems);
/// transfiere los datos del device al host
int dataTransferToHost(int *hst_result, int *hst_vector2, int *dev_result, int *dev_vector2, int numberOfItems );
/// muestra por pantalla los datos del host
int printData(int *hst_vector1, int *hst_vector2, int *hst_result, int numOfItems);
/// función que muestra por pantalla la salida del programa
int getAppOutput();

int main() {
    int deviceCount;
    int numberOfItems = 0;
    int blocks = 0;
    int *hst_vector1,*hst_vector2,*hst_result;
    int *dev_vector1,*dev_vector2,*dev_result;

    /// buscando dispositivos
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        /// mostramos el error si no se encuentra un dispositivo
        return getErrorDevice();
    } else {
        /// mostramos los datos de los dispositivos CUDA encontrados
        getDataDevice(deviceCount);
    }
    /// solicitamos al usuario la cantidad de elementos
    requestNumberOfItems(&numberOfItems);
    /// reserva del espacio de memoria en el host
    hst_vector1 = (int*)malloc( numberOfItems * sizeof(int) );
    hst_vector2 = (int*)malloc( numberOfItems * sizeof(int) );
    hst_result = (int*)malloc( numberOfItems * sizeof(int) );
    /// reserva del espacio de memoria en el device
    hipMalloc( (void**)&dev_vector1, numberOfItems * sizeof(int) );
    hipMalloc( (void**)&dev_vector2, numberOfItems * sizeof(int) );
    hipMalloc( (void**)&dev_result, numberOfItems * sizeof(int) );
    /// cargamos los datos iniciales en el host
    loadHostData(hst_vector1, hst_vector2, numberOfItems);
    /// transferimos los datos del host al device
    dataTransferToDevice(hst_vector1, dev_vector1, numberOfItems);
    /// calculamos el número de bloques a lanzar
    blocks = (int) numberOfItems/MAX_THREADS_PER_BLOCK + 1;
    /// mostramos los datos con los que llamamos al device
    printf("Lanzamiento de: %d bloques y %d hilos (%d hilos) \n", blocks, MAX_THREADS_PER_BLOCK, blocks*MAX_THREADS_PER_BLOCK);
    /// sumamos los items
    suma<<< blocks, MAX_THREADS_PER_BLOCK >>>(dev_vector1, dev_vector2, dev_result, numberOfItems);
    /// transferimos los datos del device al host
    dataTransferToHost(hst_result,hst_vector2,dev_result,dev_vector2,numberOfItems);
    /// muestra por pantalla los datos del host
    printData(hst_vector1,hst_vector2,hst_result,numberOfItems);
    /// función que muestra por pantalla la salida del programa
    getAppOutput();
    /// liberamos los recursos del device
    hipFree(dev_vector1);
    hipFree(dev_vector2);
    hipFree(dev_result);
    return 0;
}

int getErrorDevice() {
    printf("¡No se ha encontrado un dispositivo CUDA!\n");
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 1;
}

int getDataDevice(int deviceCount) {
    printf("Se han encontrado %d dispositivos CUDA:\n", deviceCount);
    for (int deviceID = 0; deviceID < deviceCount; deviceID++) {
        ///obtenemos las propiedades del dispositivo CUDA
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, deviceID);
        getDeviceProperties(deviceID, getCudaCores(deviceProp), deviceProp);
    }
    return 0;
}

int getCudaCores(hipDeviceProp_t deviceProperties) {
    int cudaCores = 0;
    int major = deviceProperties.major;
    if (major == 1) {
        /// TESLA
        cudaCores = 8;
    } else if (major == 2) {
        /// FERMI
        if (deviceProperties.minor == 0) {
            cudaCores = 32;
        } else {
            cudaCores = 48;
        }
    } else if (major == 3) {
        /// KEPLER
        cudaCores = 192;
    } else if (major == 5) {
        /// MAXWELL
        cudaCores = 128;
    } else if (major == 6 || major == 7 || major == 8) {
        /// PASCAL, VOLTA (7.0), TURING (7.5), AMPERE
        cudaCores = 64;
    } else {
        /// ARQUITECTURA DESCONOCIDA
        cudaCores = 0;
        printf("¡Dispositivo desconocido!\n");
    }
    return cudaCores;
}

int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties) {
    int SM = cudaProperties.multiProcessorCount;
    printf("***************************************************\n");
    printf("DEVICE %d: %s\n", deviceId, cudaProperties.name);
    printf("***************************************************\n");
    printf("- Capacidad de Computo            \t: %d.%d\n", cudaProperties.major, cudaProperties.minor);
    printf("- No. de MultiProcesadores        \t: %d \n", SM);
    printf("- No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores * SM);
    printf("- Memoria Global (total)          \t: %zu MiB\n", cudaProperties.totalGlobalMem / MB);
    printf("- No. maximo de Hilos (por bloque)\t: %d\n", cudaProperties.maxThreadsPerBlock);
    printf("  > [Eje x -> %d]\n", cudaProperties.maxThreadsDim[0]);
    printf("  > [Eje y -> %d]\n", cudaProperties.maxThreadsDim[1]);
    printf("  > [Eje z -> %d]\n", cudaProperties.maxThreadsDim[2]);
    printf("- No. maximo de Bloques (por eje)\n");
    printf("  > [Eje x -> %d]\n", cudaProperties.maxGridSize[0]);
    printf("  > [Eje y -> %d]\n", cudaProperties.maxGridSize[1]);
    printf("  > [Eje z -> %d]\n", cudaProperties.maxGridSize[2]);
    printf("***************************************************\n");
    return 0;
}

int requestNumberOfItems(int *numberOfItems) {
    int status = 0;
    while (status == 0) {
        printf("Introduce el numero de elementos: \n");
        scanf_s("%d", numberOfItems);
        if (MAX_INT_NUMBER < *numberOfItems || *numberOfItems < 0) {
            printf("Numero maximo de elementos superado: %d \n", MAX_INT_NUMBER);
        } else {
            printf("El numero de elementos elegido es: %d \n", *numberOfItems);
            status = 1;
        }
    }
    return 0;
}

int loadHostData(int *hst_vector1, int *hst_vector2, int numberOfItems) {
    srand ( (int)time(nullptr) );
    for (int i=0; i<numberOfItems; i++)  {
        /// inicializamos hst_vector1 con numeros aleatorios entre 0 y 1
        hst_vector1[i] = (int) rand() % 10;
        /// inicializamos hst_vector2 con ceros
        hst_vector2[i] = 0;
    }
    return 0;
}

int dataTransferToDevice(int *hst_vector1, int *dev_vector1, int numberOfItems ) {
    /// transfiere datos de hst_A a dev_A
    hipMemcpy(dev_vector1,hst_vector1, numberOfItems * sizeof(int),hipMemcpyHostToDevice);
    return 0;
}

__global__ void suma(const int *dev_vector1, int *dev_vector2, int *dev_result, int numberOfItems) {
    /// identificador del hilo
    unsigned int idGlobal = threadIdx.x + blockDim.x * blockIdx.x;
    if (numberOfItems > idGlobal) {
        /// inicializamos el vector 2
        dev_vector2[idGlobal] = dev_vector1[numberOfItems - idGlobal - 1];
        /// sumamos los dos vectores y escribimos el resultado
        dev_result[idGlobal] = dev_vector1[idGlobal] + dev_vector2[idGlobal];
    }
}

int dataTransferToHost(int *hst_result, int *hst_vector2, int *dev_result, int *dev_vector2, int numberOfItems ) {
    /// transfiere datos de dev_vector2 a hst_vector2
    hipMemcpy(hst_vector2, dev_vector2, numberOfItems * sizeof(int), hipMemcpyDeviceToHost);
    /// transfiere datos de dev_result a hst_result
    hipMemcpy(hst_result,dev_result,numberOfItems * sizeof(int),hipMemcpyDeviceToHost);
    return 0;
}

int printData(int *hst_vector1, int *hst_vector2, int *hst_result, int numOfItems) {
    printf("VECTOR 1:\n");
    for (int i = 0; i < numOfItems; i++)  {
        printf("%d ", hst_vector1[i]);
    }
    printf("\n");
    printf("VECTOR 2:\n");
    for (int i = 0; i < numOfItems; i++)  {
        printf("%d ", hst_vector2[i]);
    }
    printf("\n");
    printf("RESULTADO:\n");
    for (int i = 0; i < numOfItems; i++)  {
        printf("%d ", hst_result[i]);
    }
    printf("\n");
    return 0;
}

int getAppOutput() {
    /// salida del programa
    time_t fecha;
    time(&fecha);
    printf("***************************************************\n");
    printf("Programa ejecutado el: %s", ctime(&fecha));
    printf("***************************************************\n");
    /// capturamos un INTRO para que no se cierre la consola de MSVS
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}
